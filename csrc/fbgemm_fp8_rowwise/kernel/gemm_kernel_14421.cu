
#include <hip/hip_runtime.h>
#include "../fp8_gemm_common_hip.cuh"
#include "../fp8_gemm_common_splitk_hip.cuh"

constexpr uint32_t BLOCKS_X = 1;
constexpr uint32_t BLOCKS_Y = 4;
constexpr uint32_t BLOCKS_Z = 4;
constexpr uint32_t MBLOCKS_X = 2;
constexpr uint32_t MBLOCKS_Y = 1;

at::Tensor f8f8bf16_rowwise_32x32x16_14421(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_32x32x16::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_32x32x16::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 16x");
            LAUNCH_KERNEL_OUTTYPE_32x32x16(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            LAUNCH_KERNEL_OUTTYPE_16x16x32(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421_sk128(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32_splitK::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            TORCH_CHECK(custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z <= 128, "K chunk size is too small to split workload along K");
            LAUNCH_KERNEL_OUTTYPE_16x16x32_SK(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, 128, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421_sk256(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32_splitK::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            TORCH_CHECK(custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z <= 256, "K chunk size is too small to split workload along K");
            LAUNCH_KERNEL_OUTTYPE_16x16x32_SK(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, 256, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421_sk512(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32_splitK::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            TORCH_CHECK(custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z <= 512, "K chunk size is too small to split workload along K");
            LAUNCH_KERNEL_OUTTYPE_16x16x32_SK(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, 512, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421_sk1024(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32_splitK::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            TORCH_CHECK(custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z <= 1024, "K chunk size is too small to split workload along K");
            LAUNCH_KERNEL_OUTTYPE_16x16x32_SK(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, 1024, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_14421_sk2048(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32_splitK::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            TORCH_CHECK(custom_fp8_16x16x32_splitK::BLOCK_K * BLOCKS_Z <= 2048, "K chunk size is too small to split workload along K");
            LAUNCH_KERNEL_OUTTYPE_16x16x32_SK(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, 2048, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}
