
#include <hip/hip_runtime.h>
#include "../fp8_gemm_common_hip.cuh"

constexpr uint32_t BLOCKS_X = 1;
constexpr uint32_t BLOCKS_Y = 1;
constexpr uint32_t BLOCKS_Z = 1;
constexpr uint32_t MBLOCKS_X = 1;
constexpr uint32_t MBLOCKS_Y = 4;

at::Tensor f8f8bf16_rowwise_32x32x16_11114(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_32x32x16::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_32x32x16::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 16x");
            LAUNCH_KERNEL_OUTTYPE_32x32x16(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}

at::Tensor f8f8bf16_rowwise_16x16x32_11114(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // Not implemented
    bool use_fast_accum, // Not implemented
    std::optional<at::ScalarType> out_dtype
) {
    const at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8_16x16x32::f8f8bf16_rowwise_wrapper(
        [_out_dtype](at::Tensor XQ, at::Tensor WQ, at::Tensor x_scale, at::Tensor w_scale, at::Tensor Y, int M, int N, int K) -> void {
            TORCH_CHECK(K % (custom_fp8_16x16x32::BLOCK_K * BLOCKS_Z) == 0, "K must be divisible by 32x");
            LAUNCH_KERNEL_OUTTYPE_16x16x32(_out_dtype, BLOCKS_X, BLOCKS_Y, BLOCKS_Z, MBLOCKS_X, MBLOCKS_Y, M, N, K)
        },
        XQ, WQ, x_scale, w_scale, use_fast_accum, _out_dtype
    );
}
