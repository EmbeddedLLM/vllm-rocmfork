#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

#include "cuda_compat.h"

namespace custom_fp8 {

constexpr uint32_t BLOCK_M = 32;
constexpr uint32_t BLOCK_N = 32;
constexpr uint32_t BLOCK_K = 16;
constexpr uint32_t BLOCKS_X = 2;
constexpr uint32_t BLOCKS_Y = 2;
constexpr uint32_t BLOCKS_Z = 2;
constexpr uint32_t MBLOCKS_X = 2;
constexpr uint32_t MBLOCKS_Y = 2;

constexpr uint32_t TBLOCKS_M = BLOCK_M * BLOCKS_X;
constexpr uint32_t TBLOCKS_N = BLOCK_N * BLOCKS_Y;
constexpr uint32_t TBLOCKS_K = BLOCK_K * BLOCKS_Z;
constexpr uint32_t MBLOCKS_M = TBLOCKS_M * MBLOCKS_X;
constexpr uint32_t MBLOCKS_N = TBLOCKS_N * MBLOCKS_Y;

constexpr uint32_t LAUNCH_WARP_SIZE = 64;

// template <typename T>
// __host__ __device__ inline T ceildiv(T a, T b) { return (a + b - 1) / b; }

#define ceildiv(a, b) (((a) + (b) - 1) / (b))

enum class MatrixType {
    A = 0,
    B = 1,
};

__device__ inline void initialize_smem(uint32_t* smem, uint32_t size) {
    uint32_t available_threads = blockDim.x * blockDim.y * blockDim.z;
    uint32_t num_iters = ceildiv(size, available_threads);
    for (int i = 0; i < num_iters; ++i) {
        uint32_t id = threadIdx.z * (blockDim.y * blockDim.x) + threadIdx.y * blockDim.x + threadIdx.x;
        if (id < size) {
            smem[id] = 0x0;
        }
    }
}

template <typename T>
__device__ inline void swap_ptr(T* &a, T* &b) {
    T* tmp = a;
    a = b;
    b = tmp;
}

template <typename T>
__device__ inline void load_fp8_gds_to_lds_warp_32x32_packed4u32(
    uint32_t* const lds, const T * const gds, // indexed at warp level
    uint32_t M, uint32_t K,
    uint32_t M_index_warp, uint32_t K_index_warp
) {
    // At the warp level
    using _int4 = __attribute__((__vector_size__(4 * sizeof(uint32_t)))) uint32_t;
    const T* gds_head_elem = gds + (threadIdx.x / 2) * K;
    const _int4* gds_packed = reinterpret_cast<const _int4*>(gds_head_elem);
    gds_packed += (threadIdx.x % 2);

    bool K_within_bound = (K_index_warp + BLOCK_K * (threadIdx.x % 2 + 1)) <= K;
    bool M_within_bound = (M_index_warp + threadIdx.x / 2) < M;

    _int4* lds_packed = reinterpret_cast<_int4*>(lds);
    // lds_packed += (threadIdx.x / 2) * 2 + (threadIdx.x % 2); // threadIdx.x;
    lds_packed += threadIdx.x;

    __syncthreads();

    if (K_within_bound && M_within_bound) {
        *lds_packed = *gds_packed;
    } else {
        *lds_packed = {0, 0, 0, 0};
    }
}

// For loading only
template <MatrixType TMat>
__device__ inline uint32_t* get_lds_load_warp_head_addr(uint32_t* const lds) {
    constexpr uint32_t lds_warp_addr_leap_u32 = 
        (TMat == MatrixType::A) ? BLOCK_M * TBLOCKS_K / 4 : BLOCK_N * TBLOCKS_K / 4;
    uint32_t* lds_warp_addr_head = lds;
    if constexpr (TMat == MatrixType::A) {
        lds_warp_addr_head += lds_warp_addr_leap_u32 * (threadIdx.z + threadIdx.y * BLOCKS_X);
    } else {
        lds_warp_addr_head += lds_warp_addr_leap_u32 * (threadIdx.z + threadIdx.y * BLOCKS_Y);
    }
    return lds_warp_addr_head;
}

template <typename T, MatrixType TMat>
__device__ inline void load_fp8_gds_to_lds_tb_128x32_packed4u32(
    uint32_t* const lds, const T * const gds, // indexed at global level
    uint32_t M, uint32_t K,
    uint32_t M_index_tb, uint32_t K_index_tb
) {
    // At the threadblock level
    uint32_t* lds_warp_addr_head = get_lds_load_warp_head_addr<TMat>(lds);
    
    const uint32_t M_index_warp_head = M_index_tb + threadIdx.y * ((TMat == MatrixType::A) ? TBLOCKS_M : TBLOCKS_N);
    const uint32_t M_index_warp_gds_load = M_index_warp_head + threadIdx.z * ((TMat == MatrixType::A) ? BLOCK_M : BLOCK_N);

    const T* gds_warp_head = gds + M_index_warp_gds_load * K + K_index_tb;
    
    load_fp8_gds_to_lds_warp_32x32_packed4u32<T>(lds_warp_addr_head, gds_warp_head, M, K, M_index_warp_gds_load, K_index_tb);
}

template <int X = BLOCKS_X, int Y = BLOCKS_Y, int Z = 16>
__device__ inline uint32_t get_acc_index(uint32_t i, uint32_t j, uint32_t k = 0) {
    return i * Y * Z + j * Z + k;
}

// Given the lane and gpr ID, return the offset from the warp-level smem head of the 32-bit content
//   to be loaded onto the gpr. smem addresses are indexed in strides of u32
template <MatrixType MatT>
__device__ inline uint32_t get_smem_element_offset_warp_32x16_u32(
    uint32_t lane, uint32_t gpr_num, uint32_t smem_row_stride
) {
    if constexpr (MatT == MatrixType::A) {
        // A matrix
        uint32_t i = lane % 32;
        uint32_t k = 2 * (lane / 32) + gpr_num;
        return i * smem_row_stride + k;
    } else {
        // B matrix
        uint32_t i = lane % 32;
        uint32_t k = 2 * (lane / 32) + gpr_num;
        return i * smem_row_stride + k;
    }
}

__device__ inline void mfma_f32_32x32x16_fp8_fp8(
    const uint32_t * const A_warp_head, // indexed at warp level
    const uint32_t * const B_warp_head, // indexed at warp level
    float* acc_block,
    uint32_t A_row_stride,
    uint32_t B_row_stride
) {
    using floatx16 = __attribute__((__vector_size__(16 * sizeof(float)))) float;
    using uint32x2 = __attribute__((__vector_size__(2 * sizeof(uint32_t)))) uint32_t;
    union _reg_load {
        uint32x2 regs_;
        long long_;
    };
    // warp-level workers
    for (uint32_t k_inner_iter = 0; k_inner_iter < BLOCKS_Z; ++k_inner_iter) {
        const uint32_t* A_warp_head_inner = A_warp_head + k_inner_iter * (BLOCK_K / 4);
        const uint32_t* B_warp_head_inner = B_warp_head + k_inner_iter * (BLOCK_K / 4);
        _reg_load a_regs;
        _reg_load b_regs;
        for (int reg = 0; reg < 2; ++reg) {
            a_regs.regs_[reg] = A_warp_head_inner[get_smem_element_offset_warp_32x16_u32<MatrixType::A>(threadIdx.x, reg, A_row_stride)];
            b_regs.regs_[reg] = B_warp_head_inner[get_smem_element_offset_warp_32x16_u32<MatrixType::B>(threadIdx.x, reg, B_row_stride)];
        }
        floatx16* acc_block_f16 = reinterpret_cast<floatx16*>(acc_block);
        *acc_block_f16 = __builtin_amdgcn_mfma_f32_32x32x16_fp8_fp8(a_regs.long_, b_regs.long_, *acc_block_f16, 0, 0, 0);
    }
}

__device__ inline void mfma_f32_64x64x32_fp8_fp8(
    const uint32_t * const A_head_tb, // indexed at threadblock level
    const uint32_t * const B_head_tb, // indexed at threadblock level
    float* acc, // blocksX x blocksY x 16,
    uint32_t A_row_stride, // in u32
    uint32_t B_row_stride  // in u32 (BLOCK_K / 4) * BLOCKS_Z
) {
    uint32_t warp_x = threadIdx.y;
    uint32_t warp_y = threadIdx.z;
    for (uint32_t x_iter = 0; x_iter < BLOCKS_X; ++x_iter) {
        uint32_t x_load_block = warp_x * BLOCKS_X + x_iter;
        const uint32_t* A_warp_head = A_head_tb + x_load_block * BLOCK_M * A_row_stride;
        for (uint32_t y_iter = 0; y_iter < BLOCKS_Y; ++y_iter) {
            uint32_t y_load_block = warp_y * BLOCKS_Y + y_iter;
            const uint32_t* B_warp_head = B_head_tb + y_load_block * BLOCK_N * B_row_stride;
            float* acc_block = acc + get_acc_index(x_iter, y_iter);
            mfma_f32_32x32x16_fp8_fp8(A_warp_head, B_warp_head, acc_block, A_row_stride, B_row_stride);
        }
    }
}

template <typename TF32, MatrixType MatT>
__device__ inline void load_scale_gds_to_lds_128(
    const TF32 * const gds, // indexed at global level
    float* lds,
    uint32_t tb_head_index,
    uint32_t size
) {
    const TF32* gds_head_tb = gds + tb_head_index;
    bool active = false;
    uint32_t offset = 0;
    uint32_t num_iters = 1;
    if constexpr (MatT == MatrixType::A) {
        offset = threadIdx.y * TBLOCKS_M + threadIdx.x;
        num_iters = TBLOCKS_M / LAUNCH_WARP_SIZE;
        active = !(threadIdx.z % BLOCKS_Y) && (offset + num_iters - 1 < size);
    } else {
        offset = threadIdx.z * TBLOCKS_N + threadIdx.x;
        num_iters = TBLOCKS_N / LAUNCH_WARP_SIZE;
        active = !(threadIdx.y % BLOCKS_X) && (offset + num_iters - 1 < size);
    }

    __syncthreads();
    if (active) {
#pragma unroll
        for (uint32_t iter = 0; iter < num_iters; ++iter) {
            uint32_t offset_iter = offset + iter;
            lds[offset_iter] = (offset_iter < size) ? static_cast<float>(gds_head_tb[offset + iter]) : 0.0f;
        }
    }
    __syncthreads();
}

template <typename TF32>
__device__ inline void apply_scale(
    const float * const lds_xscale,
    const TF32 * const gds_wscale, // Indexed at global level
    float* acc,
    uint32_t wscale_tb_head_index,
    uint32_t wscale_size
) {
    // Each warp applies scales to a 64x64 block
    const uint32_t lane_id = threadIdx.x;
    uint32_t gds_wscale_warp_offset = wscale_tb_head_index + threadIdx.z * BLOCKS_Y * BLOCK_N;
    const float* lds_xscale_warp = lds_xscale + threadIdx.y * BLOCKS_X * BLOCK_M;
#pragma unroll
    for (uint32_t warp_y = 0; warp_y < BLOCKS_Y; ++warp_y) {
        uint32_t gds_wscale_warp_iter_offset = gds_wscale_warp_offset + warp_y * BLOCK_N;
        uint32_t wscale_offset_thread = gds_wscale_warp_iter_offset + (lane_id % 32);
        __syncthreads();
        float wscale = (wscale_offset_thread < wscale_size) ? 
            static_cast<float>(gds_wscale[wscale_offset_thread]) : 0.0f;

        for (uint32_t warp_x = 0; warp_x < BLOCKS_X; ++warp_x) {
            float* acc_warp = acc + get_acc_index(warp_x, warp_y);
            const float* lds_xscale_warp_iter = lds_xscale_warp + warp_x * BLOCK_M;

#pragma unroll
            for (uint32_t reg = 0; reg < 16; ++reg) {
                acc_warp[reg] *= wscale;
                acc_warp[reg] *= lds_xscale_warp_iter[(8 * (reg / 4) % 32) + 4 * (lane_id / 32) + (reg % 4)];
            }
        }
    }
}

template <typename TY>
__device__ inline void store_acc_to_gds_transposed(
    TY* y_gds,
    float* acc,
    uint32_t M_head_index, // col head id
    uint32_t N_head_index, // row head id
    uint32_t M,
    uint32_t N,
    uint32_t y_row_stride // should be M
) {
    // constexpr uint32_t pack_factor = 4; // Must be 4
    const uint32_t lane_id = threadIdx.x;
    const uint32_t M_warp_head_offset = M_head_index + threadIdx.y * TBLOCKS_M;
    const uint32_t N_warp_head_offset = N_head_index + threadIdx.z * TBLOCKS_N;
    for (uint32_t warp_m = 0; warp_m < BLOCKS_X; ++warp_m) {
        uint32_t M_warp_iter_head_offset = M_warp_head_offset + warp_m * BLOCK_M;
        for (uint32_t warp_n = 0; warp_n < BLOCKS_Y; ++warp_n) {
            uint32_t N_warp_iter_head_offset = N_warp_head_offset + warp_n * BLOCK_N;
            uint32_t N_lane_reg_offset = N_warp_iter_head_offset + (lane_id % 32);
            if (N_lane_reg_offset >= N) { continue; }
            uint32_t N_offset_strided = N_lane_reg_offset * y_row_stride;

            for (uint32_t reg = 0; reg < 16; reg += 4) {
                uint32_t M_lane_reg_offset = M_warp_iter_head_offset + (8 * (reg / 4) % 32) + 4 * (lane_id / 32) + (reg % 4);
                if (M_lane_reg_offset >= M) { continue; }
                if constexpr (sizeof(TY) == 2) {
                    using TY4 = __attribute__((__vector_size__(4 * 2))) uint16_t;
                    TY buffer[4];
                    uint32_t acc_index_base = get_acc_index(warp_m, warp_n);
                    for (int rr = 0; rr < 4; ++rr) {
                        buffer[rr] = static_cast<TY>(acc[get_acc_index(warp_m, warp_n) + reg + rr]);
                    }
                    *(reinterpret_cast<TY4 *>(y_gds + N_offset_strided + M_lane_reg_offset)) = *(reinterpret_cast<TY4 *>(buffer));
                } else {
                    *(reinterpret_cast<int4 *>(y_gds + N_offset_strided + M_lane_reg_offset)) = 
                        *(reinterpret_cast<int4 *>(acc + get_acc_index(warp_m, warp_n, reg)));
                }
            }

        }
    }
}

template <typename TF8, typename TF32, typename TY>
__global__ void f8f8f16_rowwise_kernel(
    const TF8 * const xq,
    const TF8 * const wq,
    const TF32 * const x_scale,
    const TF32 * const w_scale,
    TY* y,
    uint32_t M,
    uint32_t N,
    uint32_t K
) {
    constexpr uint32_t A_block_size_u32 = BLOCK_M * BLOCK_K / (4 / sizeof(TF8));
    constexpr uint32_t A_warp_block_size_u32 = A_block_size_u32 * BLOCKS_X * BLOCKS_Z;
    constexpr uint32_t A_tile_block_size_u32 = A_warp_block_size_u32 * MBLOCKS_X;
    constexpr uint32_t B_block_size_u32 = BLOCK_N * BLOCK_K / (4 / sizeof(TF8));
    constexpr uint32_t B_warp_block_size_u32 = B_block_size_u32 * BLOCKS_Y * BLOCKS_Z;
    constexpr uint32_t B_tile_block_size_u32 = B_warp_block_size_u32 * MBLOCKS_Y;

    __shared__ uint32_t A_shared[A_tile_block_size_u32 * 2];
    __shared__ uint32_t B_shared[B_tile_block_size_u32 * 2]; // transposed

    constexpr uint32_t A_row_stride = BLOCK_K / (4 / sizeof(TF8)) * BLOCKS_Z;
    constexpr uint32_t B_row_stride = BLOCK_K / (4 / sizeof(TF8)) * BLOCKS_Z;

    uint32_t* A_shared_load = A_shared;
    uint32_t* A_shared_eval = A_shared + A_tile_block_size_u32;
    uint32_t* B_shared_load = B_shared;
    uint32_t* B_shared_eval = B_shared + A_tile_block_size_u32;

    initialize_smem(A_shared_load, A_tile_block_size_u32);
    initialize_smem(B_shared_load, B_tile_block_size_u32);

    float acc[BLOCKS_X * BLOCKS_Y * 16];
    for (uint32_t i = 0; i < BLOCKS_X * BLOCKS_Y * 16; ++i) {
        acc[i] = 0.0f;
    }

    const uint32_t M_index_tile = blockIdx.x * MBLOCKS_M; // head of threadblock
    const uint32_t N_index_tile = blockIdx.y * MBLOCKS_N; // head of threadblock

    const uint32_t k_iters = ceildiv(K, BLOCK_K * BLOCKS_Z);
    uint32_t K_index_tile = 0;

    __syncthreads();

    // Iteration #0 loading
    load_fp8_gds_to_lds_tb_128x32_packed4u32<TF8, MatrixType::A>(A_shared_load, xq, M, K, M_index_tile, K_index_tile);
    load_fp8_gds_to_lds_tb_128x32_packed4u32<TF8, MatrixType::B>(B_shared_load, wq, N, K, N_index_tile, K_index_tile);

    swap_ptr(A_shared_load, A_shared_eval);
    swap_ptr(B_shared_load, B_shared_eval);

    __syncthreads();

    for (int kk = 1; kk < k_iters; ++kk) {
        // load
        K_index_tile += BLOCK_K * BLOCKS_Z;
        load_fp8_gds_to_lds_tb_128x32_packed4u32<TF8, MatrixType::A>(A_shared_load, xq, M, K, M_index_tile, K_index_tile);
        load_fp8_gds_to_lds_tb_128x32_packed4u32<TF8, MatrixType::B>(B_shared_load, wq, N, K, N_index_tile, K_index_tile);

        // compute mm
        mfma_f32_64x64x32_fp8_fp8(A_shared_eval, B_shared_eval, acc, A_row_stride, B_row_stride);

        // swap
        swap_ptr(A_shared_load, A_shared_eval);
        swap_ptr(B_shared_load, B_shared_eval);

        __syncthreads();
    }
    // Iteration #-1 computing
    mfma_f32_64x64x32_fp8_fp8(A_shared_eval, B_shared_eval, acc, A_row_stride, B_row_stride);

    // Apply scales
    float* x_scale_shared = reinterpret_cast<float*>(A_shared_load);
    load_scale_gds_to_lds_128<TF32, MatrixType::A>(x_scale, x_scale_shared, M_index_tile, M);
    apply_scale(x_scale_shared, w_scale, acc, N_index_tile, N);
    __syncthreads();

    // Save the result to gds in transpose to facilitate coalescence
    store_acc_to_gds_transposed(y, acc, M_index_tile, N_index_tile, M, N, M);
}

at::Tensor f8f8bf16_rowwise_impl(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    at::Tensor Y
) {
    int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
    int N = WQ.size(0);
    int K = WQ.size(1);

    dim3 grid(ceildiv(M, MBLOCKS_M), ceildiv(N, MBLOCKS_N), 1);
    dim3 block(LAUNCH_WARP_SIZE, MBLOCKS_X, MBLOCKS_Y);

    auto stream{torch::hip::getCurrentHIPStream().stream()};

#define LAUNCH_KERNEL(TFY) \
        { \
        auto kernel = f8f8f16_rowwise_kernel<uint8_t, float, TFY>; \
        kernel<<<grid, block, 0, stream>>>( \
            reinterpret_cast<uint8_t*>(XQ.data_ptr()), \
            reinterpret_cast<uint8_t*>(WQ.data_ptr()), \
            reinterpret_cast<float*>(x_scale.data_ptr()), \
            reinterpret_cast<float*>(w_scale.data_ptr()), \
            reinterpret_cast<TFY*>(Y.data_ptr()), \
            M, N, K \
        ); \
        }

    if (Y.dtype() == at::kFloat) {
        LAUNCH_KERNEL(float)
    } else if (Y.dtype() == at::kHalf) {
        LAUNCH_KERNEL(__half)
    } else if (Y.dtype() == at::kBFloat16) {
        LAUNCH_KERNEL(__hip_bfloat16)
    } else {
        AT_ERROR("Not implemented output datatype. Must be one of {float, half, bfloat16}.");
    }
    
    return Y;
}

at::Tensor f8f8bf16_rowwise_wrapper(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias, // not calculated
    bool use_fast_accum,
    std::optional<at::Tensor> output = std::nullopt,
    std::optional<at::ScalarType> out_dtype = std::nullopt) {
    // Check that input datatypes are valid.
    TORCH_CHECK(
        (XQ.dtype() == at::kFloat8_e4m3fnuz) &&
            (WQ.dtype() == at::kFloat8_e4m3fnuz),
        "Inputs must be type float8_e4m3fnuz.");
    TORCH_CHECK(
        (x_scale.dtype() == at::kFloat) && (w_scale.dtype() == at::kFloat),
        "Scales must be float32.");
    TORCH_CHECK(use_fast_accum, "AMD does not support disabling use_fast_accum.");

    // Check inputs are in expected format.
    TORCH_CHECK(XQ.is_cuda() && XQ.is_contiguous());
    TORCH_CHECK(WQ.is_cuda() && WQ.is_contiguous());

    // XQ: M x K
    // WQ: N x K
    // output: M x N
    int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
    int N = WQ.size(0);
    int K = WQ.size(1);

    TORCH_CHECK((K % 16) == 0, 
        "Cases where K is not divisible by 16 has not been implemented.");

    at::ScalarType _out_dtype = (out_dtype.has_value()) ? out_dtype.value() : at::kBFloat16;

    // Prepare output tensor if needed.
    at::Tensor Y;
    if (output.has_value()) {
        Y = output.value();
        // Make sure the provided output has the proper shape and dtype.
        if (Y.dim() >= 3) {
            int B = size_to_dim_(Y.dim() - 2, Y.sizes());
            int Y_M = Y.size(Y.dim() - 1);
            int Y_N = Y.size(Y.dim() - 2);
            TORCH_CHECK(Y_M*B == M && Y_N == N, "Y must be transposed");
        } else if (Y.dim() == 2) {
            int Y_M = Y.size(Y.dim() - 1);
            int Y_N = Y.size(Y.dim() - 2);
            TORCH_CHECK(Y_M == M && Y_N == N, "Y must be transposed");
        } else {
            AT_ERROR("Output should at least have two dimensions");
        }
        TORCH_CHECK(Y.dtype() == _out_dtype);
    } else {
        // 1. If the input tensor is {M, K}, the output tensor is {M, N}.
        // 2. If the input tensor is {b, M, K}, the output tensor is {b, M, N}.
        if (XQ.dim() >= 3) {
            int B = size_to_dim_(XQ.dim() - 2, XQ.sizes());
            int X_M = XQ.size(XQ.dim() - 2);
            int W_N = WQ.size(WQ.dim() - 1);
            Y = at::empty({B, W_N, X_M}, XQ.options().dtype(_out_dtype));
        } else if (XQ.dim() == 2) {
            int X_M = XQ.size(XQ.dim() - 2);
            int W_N = WQ.size(WQ.dim() - 2);
            Y = at::empty({W_N, X_M}, XQ.options().dtype(_out_dtype));
        } else {
            AT_ERROR("Output should at least have two dimensions");
        }
    }

    return f8f8bf16_rowwise_impl(XQ, WQ, x_scale, w_scale, Y);
}

} // namespace custom_fp8


at::Tensor f8f8bf16_rowwise(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    bool use_fast_accum,
    std::optional<at::ScalarType> out_dtype
) {
    // Invoke f8f8bf16 rowwise without preallocated output.
    return custom_fp8::f8f8bf16_rowwise_wrapper(
        XQ, WQ, x_scale, w_scale, bias, use_fast_accum, std::nullopt, out_dtype);
}
