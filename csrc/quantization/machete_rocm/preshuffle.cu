#include "hip/hip_runtime.h"
#include "ops.cuh"
#include <iomanip>
#include <iostream>
#include <typeinfo>
#include <functional>
#include <ATen/ATen.h>
#ifdef USE_ROCM
#include <c10/hip/HIPStream.h>
#else
#include <c10/cuda/CUDAStream.h>
#endif
#include "ck/tensor_operation/gpu/device/tensor_layout.hpp"
#include "ck/library/utility/device_memory.hpp"
#include "ck/library/utility/host_tensor.hpp"
#include "ck/library/utility/literals.hpp"
#include "utils.hpp"

#define KPack 16
#define KLane 64 / NLane
#define ThreadsPerBlock 256

using FP8  = ck::f8_t;

namespace {

  int __host__ __device__ compute_lane_index(
    const int n,
    const int k,
    const int K,
    const int NXdl
  ){
    // K -> K0 KLane KPack
    // N -> N0 NLane
    // N, K -> N0 K0 KLane NLane KPack
    const int NLane = NXdl;
    const int K0 = K / (KLane * KPack);
    const int n0 = n / NLane;
    const int n1 = n % NLane;
    const int k0 = k / (KLane * KPack);
    const int tempk = k % (KLane * KPack);
    const int k1 = tempk / KPack;
    const int k2 = tempk % KPack;

    return n0 * KPack * NLane * KLane * K0 + 
            k0 * KPack * NLane * KLane      +
            k1 * KPack * NLane              +
            n1 * KPack                      + 
            k2;
  }

  template <typename scalar_t>
  static __global__ void preshuffle_kernel(
    const scalar_t* source,
    scalar_t* destination,
    const int N,
    const int K,
    const int NXdl
  ){
      const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
      const int total_threads = gridDim.x * blockDim.x;
      for (int i = thread_idx; i < N * K; i += total_threads) {
        int n = i / K;
        int k = i % K;
        if(n < N && k<K)
          destination[compute_lane_index(n, k, K, NXdl)] = source[n * K + k];
      }
  }

  void preshuffle_cpu_launcher(
    const FP8* source_ptr,
    FP8* destination_ptr,
    const int N, 
    const int K,
    const int NXdl
  ){
      for(int n = 0; n < N; ++n)
        for(int k = 0; k < K; ++k)
          destination_ptr[compute_lane_index(n, k, K, NXdl)] = source_ptr[n * K + k];
  }

  void preshuffle_launcher(
    const FP8* source_ptr, 
    FP8* destination_ptr, 
    const int N, 
    const int K, 
    const int NXdl
  ){
    
    const int total_elements = N * K;
    const int blocks = (total_elements + ThreadsPerBlock - 1) / ThreadsPerBlock;
    auto stream = at::cuda::getCurrentHIPStream().stream();

    preshuffle_kernel<FP8><<<blocks, ThreadsPerBlock, 0, stream>>>(
      source_ptr,
      destination_ptr,
      N,
      K,
      NXdl
    );

    const auto cuda_last_error = hipGetLastError();
    if (hipSuccess != cuda_last_error)
      throw std::runtime_error("CUDA kernel failed : " + std::to_string(cuda_last_error));
              
  }

  at::Tensor preshuffle_impl(
    const at::Tensor& source,
    const int NXdl,
    const std::function<void(const FP8*, FP8*, const int, const int, const int)>& launcher
  ){
    const auto source_aligned = machete_rocm::utils::align_to_wavefront(source);
    const FP8* source_ptr = reinterpret_cast<FP8*>(source_aligned.data_ptr());

    const auto N = source_aligned.size(0);
    const auto K = source_aligned.size(1);
    
    auto destination = torch::zeros(source_aligned.sizes(), source_aligned.options());
    FP8* destination_ptr = reinterpret_cast<FP8*>(destination.data_ptr());

    launcher(source_ptr, destination_ptr, N, K, NXdl);
    return destination;
  }

}

at::Tensor machete_rocm::preshuffle(const at::Tensor tensor, const int64_t NXdl) {
  return preshuffle_impl(tensor, NXdl, preshuffle_launcher);
}

at::Tensor machete_rocm::preshuffle_cpu(const at::Tensor tensor, const int64_t NXdl) {
  return preshuffle_impl(tensor, NXdl, preshuffle_cpu_launcher);
}

